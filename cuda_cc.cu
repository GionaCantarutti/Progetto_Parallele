#include "hip/hip_runtime.h"
#include "char_matrix.h"

#define ChunkSize 32

static void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "ERRORE CUDA: >%s<: >%s<. Eseguo: EXIT\n", msg, hipGetErrorString(err) );
        exit(-1);
    }
}


__global__ void cuda_cc(int** groups, char** mat, int width, int height) {

    int gx = blockIdx.x * blockDim.x + threadIdx.x; 	//Global x index
    int gy = blockIdx.y * blockDim.y + threadIdx.y;     //Global y index
    int gl = gy * width + gx;                           //Global linearized index
    int lx = threadIdx.x; 	                            //Local x index
    int ly = threadIdx.y;                               //Local y index
    int ll = gy * ChunkSize + gx;                       //Local linearized index

    __shared__ char groupsChunk[ChunkSize][ChunkSize];

    //Init groups
    groupsChunk[ly][lx] = ll;
    __syncthreads(); //Await end of initialization
    
    
}

GroupMatrix cuda_cc(CharMatrix* mat) {

    //Initialize and allocate device memory for groups
    int** d_groups;
    HANDLE_ERROR(hipMalloc((void**)d_groups, mat->height * mat->width * sizeof(int)));

    //Initialize and allocate device memory for character matrix
    char** d_mat;
    HANDLE_ERROR(hipMalloc((void**)d_mat, mat->height * mat->width * sizeof(char)));

    //Copy char matrix to device memory
    HANDLE_ERROR(hipMemcpy(d_mat, (void*)mat->matrix, mat->width * mat->height * sizeof(char), hipMemcpyHostToDevice));

    dim3 numBlocks(ceil(mat->width / ChunkSize), (mat->height) / ChunkSize);
    dim3 numThreads(ChunkSize, ChunkSize);
    cuda_cc<<<numBlocks, numThreads>>>(d_groups, d_mat, mat->width, mat->height);
    hipDeviceSynchronize();

    checkCUDAError("call of cuda_cc kernel");
    
    //Copy group matrix back to host
    GroupMatrix h_groups = simpleInitGroups(mat->width, mat->height);
    HANDLE_ERROR(hipMemcpy(h_groups.groups, (void*)d_groups, mat->width * mat->height * sizeof(int), hipMemcpyDeviceToHost));

    //Free device memory
    HANDLE_ERROR(hipFree(d_groups));
    HANDLE_ERROR(hipFree(d_mat));

    return h_groups;

}