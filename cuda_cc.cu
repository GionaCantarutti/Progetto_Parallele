#include "hip/hip_runtime.h"
#include "char_matrix.h"

#define ChunkSize 32    //Has to be divisible by 2

static void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "ERRORE CUDA: >%s<: >%s<. Eseguo: EXIT\n", msg, hipGetErrorString(err) );
        exit(-1);
    }
}

enum ChunkStatus {
    WAITING = 0,
    DIRTY_NORTH = 1 << 0,
    DIRTY_EAST = 1 << 1,
    DIRTY_SOUTH = 1 << 2,
    DIRTY_WEST = 1 << 3
};

//Neighbour deltas
__constant__ const int dx[] = {1, -1, 0, 0};
__constant__ const int dy[] = {0, 0, 1, -1};
//Map delta index to cardinal direction in reverse
__constant__ const int bdr[] = {1 << 3, 1 << 1, 1 << 0, 1 << 2};

__device__ void propagate(int lxc, int lyc, int gxc, int gyc, int width, int height, char* mat, int groupsChunk[ChunkSize * ChunkSize], bool* blockStable) {

    if (gxc >= width || gyc >= height) return; //Bounds check

    for (int i = 0; i < 4; i++) { //Loop over 4 neighbours
        int nlxc = lxc + dx[i]; int ngxc = gxc + dx[i];
        int nlyc = lyc + dy[i]; int ngyc = gyc + dy[i];
        if (nlxc >= ChunkSize || nlyc >= ChunkSize || nlxc < 0 || nlyc < 0) continue;   //Bounds check (local)
        if (ngxc >= width || ngyc >= height || ngxc < 0 || ngyc < 0) continue;          //Bounds check (global)

        if (mat[gyc * width + gxc] == mat[ngyc * width + ngxc]) {
            if (groupsChunk[lyc * ChunkSize + lxc] > groupsChunk[nlyc * ChunkSize + nlxc]) { //Safe thanks to chessboard pattern
                groupsChunk[lyc * ChunkSize + lxc] = groupsChunk[nlyc * ChunkSize + nlxc];
                *blockStable = false;
            }
        }
    }

}

//Propagate but can access global groups and propagate from that to the local groupsChunk
__device__ void globally_propagate(int lxc, int lyc, int gxc, int gyc, int width, int height, char* mat, int groupsChunk[ChunkSize * ChunkSize], bool* blockStable, int* groups) {

    if (gxc >= width || gyc >= height) return; //Bounds check

    for (int i = 0; i < 4; i++) { //Loop over 4 neighbours
        int ngxc = gxc + dx[i];
        int ngyc = gyc + dy[i];
        if (ngxc >= width || ngyc >= height || ngxc < 0 || ngyc < 0) continue;          //Bounds check (global)

        if (mat[gyc * width + gxc] == mat[ngyc * width + ngxc]) {
            if (groupsChunk[lyc * ChunkSize + lxc] > groups[ngyc * width + ngxc]) { //Safe thanks to chessboard pattern
                groupsChunk[lyc * ChunkSize + lxc] = groups[ngyc * width + ngxc];
                *blockStable = false;
            }
        }
    }

}

//Makes thread 0 check if there's a dirty neighbouring chunk. If so lower the corresponding directional dirty flag on that chunk
//Syncs all threads before returning
__device__ void serialCheckDirty(int ll, bool* dirtyNeighbour, ChunkStatus* status_matrix, dim3 numBlocks, int* dirtyBlocks) {
    //Thread 0 checks for dirty neighbouring blocks
    if (ll == 0) {
        for (int i = 0; i < 4; i++) {
            int nx = blockIdx.x + dx[i];    //New x value
            int ny = blockIdx.y + dy[i];    //New y value
            if (ny >= numBlocks.y || ny < 0 || nx >= numBlocks.x || nx < 0) continue; //Boundary check
            int index = ny * numBlocks.x + nx;

            int mask = ~bdr[i];

            ChunkStatus old_status = (ChunkStatus)atomicAnd((int*)&status_matrix[index], mask);
            bool was_dirty = (old_status & bdr[i]) != 0;

            if (was_dirty) {
                *dirtyNeighbour = true;
                if ((old_status & mask) == 0) { //If the last dirty bit has been cleared by this operation decrement dirtyBlocks
                    atomicAdd(dirtyBlocks, -1);
                }
            }
        }
    }
    __syncthreads();
}

//ToDo: probs there's a way to get block count without passing it as argument
__global__ void cuda_cc(int* groups, char* mat, int width, int height, ChunkStatus* status_matrix, dim3 numBlocks, int* dirtyBlocks) {

    //Each thread will handle two cells each (hence the doubled indexes). In the memory management part we split the 32x32 chunk into two 16x32 sections.
    //In the iterative algorithm part instead we split the 32x32 chunk into a chessboard pattern of alternating cells so that we can avoid race dontions.

    int blockStartX = blockIdx.x * ChunkSize;           // Each block covers 32 columns
    int blockStartY = blockIdx.y * ChunkSize;           // and 32 rows

    int lx = threadIdx.x; 	                            //Local x index
    int ly = threadIdx.y;                               //Local y index
    int ll = ly * ChunkSize + lx;                       //Local linearized index
    int lx1 = threadIdx.x + (ChunkSize/2);              //Second local x index
    int ly1 = ly;                                       //Second local y index
    int ll1 = ly1 * ChunkSize + lx1;                    //Second local linearized index
    
    int gx = blockStartX + lx; 	                        //Global x index
    int gy = blockStartY + ly;                          //Global y index
    int gl = gy * width + gx;                           //Global linearized index
    int gx1 = blockStartX + lx1; 	                    //Second Global x index
    int gy1 = blockStartY + ly1;                        //Second Global y index
    int gl1 = gy1 * width + gx1;                        //Second Global linearized index

    __shared__ int groupsChunk[ChunkSize * ChunkSize];  //Shared memory for groups of the local chunk
    __shared__ bool blockStable;                        //Is the chunk in a stable configuration?
    __shared__ bool dirtyNeighbour;                     //Are we yet to account for changes in a neighbouring chunk?
    __shared__ bool dirtyBlock;                         //Has this chunk been changed?

    bool validGlobal1 = true;                           //Is the second set of coordinates globally valid?

    //Initialize flags
    if (ll == 0) {
        blockStable = true;
        dirtyNeighbour = true;
        dirtyBlock = false;
    }

    //Bounds check
    //if (gx >= width || gy >= height) return;
    if (gx >= width || gy >= height) return;
    validGlobal1 = !(gx1 >= width || gy1 >= height);

    int big = width * height + 100;

    //Init shared memory groups
    groupsChunk[ll] = groups[gl];
    groupsChunk[ll1] = validGlobal1 ? groups[gl1] : big;


    __syncthreads(); //Await end of initialization

    /////////////////////// End of initialization ///////////////////////
    
    do {

        if (ll == 0) {
            blockStable = true;
            dirtyNeighbour = false;
        }
        //ToDo: only check this every so often?
        serialCheckDirty(ll, &dirtyNeighbour, status_matrix, numBlocks, dirtyBlocks);

        __syncthreads();

        //Chess pattern
        int lxc = lx * 2 + (ly % 2);    //Local chess x
        int gxc = blockStartX + lxc;    //Global chess x
        int lxc1 = lxc + 1;
        if (lxc1 >= ChunkSize) lxc1 -= ChunkSize; //Faster than modulo?
        int gxc1 = blockStartX + lxc1;

        if (!dirtyNeighbour) {
            propagate(lxc, ly, gxc, gy, width, height, mat, groupsChunk, &blockStable);
            __syncthreads();
            propagate(lxc1, ly, gxc1, gy, width, height, mat, groupsChunk, &blockStable);
        } else {
            globally_propagate(lxc, ly, gxc, gy, width, height, mat, groupsChunk, &blockStable, groups);
            __syncthreads();
            globally_propagate(lxc1, ly, gxc1, gy, width, height, mat, groupsChunk, &blockStable, groups);
        }

        __syncthreads(); //Sync all at the end of an iteration
        if (!blockStable) dirtyBlock = true;
        __syncthreads();
    } while (!blockStable);
    
    __threadfence();
    if (dirtyBlock) {
        //Race conditions shoulnd't be a concern here
        groups[gl] = groupsChunk[ll];   //Copy stable chunk to global
        if (validGlobal1) groups[gl1] = groupsChunk[ll1];

        __syncthreads();
        if (ll == 0) {
            // Calculate valid neighbors. If a neighbour in one direction doesn't exist the dirty flag shoulnd't be raised because nothing
            // would be able to then lower it again
            int flags = WAITING;
            if (blockIdx.y > 0)             flags |= DIRTY_NORTH;
            if (blockIdx.x < numBlocks.x-1) flags |= DIRTY_EAST;
            if (blockIdx.y < numBlocks.y-1) flags |= DIRTY_SOUTH;
            if (blockIdx.x > 0)             flags |= DIRTY_WEST;
            //Atomically check if chunk was already dirty and make it dirty
            ChunkStatus old_status = (ChunkStatus)atomicOr( (int*)&status_matrix[blockIdx.y * numBlocks.x + blockIdx.x], flags);
            if (old_status == 0) atomicAdd(dirtyBlocks, 1); // Only increment if previously clean
        }
    }
    
}

GroupMatrix cuda_cc(CharMatrix* mat) {

    dim3 numBlocks( (mat->width + ChunkSize - 1) / ChunkSize, (mat->height + ChunkSize - 1) / ChunkSize );
    dim3 numThreads(ChunkSize/2, ChunkSize);

    //Initialize and allocate device memory for groups
    int* d_groups;
    GroupMatrix h_groups = initGroupsUnique(mat->width, mat->height);
    HANDLE_ERROR(hipMalloc((void**)&d_groups, mat->height * mat->width * sizeof(int)));
    HANDLE_ERROR(hipMemcpy(d_groups, (void*)(h_groups.groups), h_groups.width * h_groups.height * sizeof(int), hipMemcpyHostToDevice));

    //Initialize and allocate device memory for character matrix
    char* d_mat;
    HANDLE_ERROR(hipMalloc((void**)&d_mat, mat->height * mat->width * sizeof(char)));

    //Copy char matrix to device memory
    HANDLE_ERROR(hipMemcpy(d_mat, (void*)mat->matrix, mat->width * mat->height * sizeof(char), hipMemcpyHostToDevice));

    //Initialize status matrix
    int statusSize = sizeof(ChunkStatus) * numBlocks.x * numBlocks.y;
    enum ChunkStatus* h_status_matrix;
    h_status_matrix = (ChunkStatus*)malloc(statusSize);
    // Initialize status matrix with by forcing a first inter-block communication before anything else happens
    for (int x = 0; x < numBlocks.x; x++) {
        for (int y = 0; y < numBlocks.y; y++) {
            int status = WAITING;
            if (y < numBlocks.y - 1) status |= DIRTY_SOUTH;
            if (x < numBlocks.x - 1) status |= DIRTY_EAST;
            if (y > 0)               status |= DIRTY_NORTH;
            if (x > 0)               status |= DIRTY_WEST;
            h_status_matrix[y * numBlocks.x + x] = (ChunkStatus)status;
        }
    }
    enum ChunkStatus* d_status_matrix;
    HANDLE_ERROR(hipMalloc((void**)&d_status_matrix, statusSize));
    HANDLE_ERROR(hipMemcpy(d_status_matrix, h_status_matrix, statusSize, hipMemcpyHostToDevice));

    //Dirty count
    int* d_dirty;
    int h_dirty = numBlocks.x * numBlocks.y; //Corner block isn't dirty to any neighbours at the start
    HANDLE_ERROR(hipMalloc((void**)&d_dirty, sizeof(int)));
    HANDLE_ERROR(hipMemcpy(d_dirty, &h_dirty, sizeof(int), hipMemcpyHostToDevice));

    int iters = 0;
    bool err = false;

    //Loop until stable
    while (h_dirty > 0 && !err) {

        //printf("Dirty blocks: %d\n", h_dirty);
        
        cuda_cc<<<numBlocks, numThreads>>>(d_groups, d_mat, mat->width, mat->height, d_status_matrix, numBlocks, d_dirty);
        HANDLE_ERROR(hipMemcpy(&h_dirty, d_dirty, sizeof(int), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
        
        checkCUDAError("call of cuda_cc kernel");

        iters++;
        if (iters > numBlocks.x * numBlocks.y * 5) {
            printf("Something went wrong! Quitting and logging solution\n");
            err = true;
        }

    }

    printf("Device is done\n");

    //Copy group matrix back to host
    HANDLE_ERROR(hipMemcpy(h_groups.groups, (void*)d_groups, mat->width * mat->height * sizeof(int), hipMemcpyDeviceToHost));

    if (err) {
        //Dump groups and dirty matrix to a file
        GroupMatrix dirtyMatrix;
        dirtyMatrix.width = numBlocks.x; dirtyMatrix.height = numBlocks.y;
        dirtyMatrix.groups = (int*)malloc(numBlocks.x * numBlocks.y * sizeof(int));
        HANDLE_ERROR(hipMemcpy(dirtyMatrix.groups, (void*)d_status_matrix, dirtyMatrix.width * dirtyMatrix.height * sizeof(int), hipMemcpyDeviceToHost));
        hipDeviceSynchronize(); //ToDo: check if needed
        saveGroupMatrixToFile(&h_groups, "Outputs/Errors/err_groups.txt");
        saveGroupMatrixToFile(&dirtyMatrix, "Outputs/Errors/err_statuses.txt");
    }

    //Free device memory
    HANDLE_ERROR(hipFree(d_groups));
    HANDLE_ERROR(hipFree(d_mat));

    hipDeviceSynchronize(); //ToDo: check if needed

    return h_groups;

}