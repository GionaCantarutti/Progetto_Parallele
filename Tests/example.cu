
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
    printf("Hello World!\n");
}

int main() {
    hello<<<1, 4>>>();

    printf("CPU says hi!\n");

    hipDeviceSynchronize();

    return 0;
}